// Test to ensure malloc-allocated memory in CUDA-enabled GLIBC is
// directly accessible from within CUDA kernel


#include <hip/hip_runtime.h>
#include <cstdio>

void __global__ kernel(int* val)
{
	printf("GPU value read: %d\n", *val);
}

#include <malloc.h>

#define CUDA_SAFE_CALL(x) \
	do { hipError_t err = x; if (err != hipSuccess) { \
		fprintf (stderr, "Error \"%s\" at %s:%d \n", hipGetErrorString(err), \
			__FILE__, __LINE__); exit(-1); \
	}} while (0);

int main(int argc, char* argv[])
{
	int* val = (int*)malloc(sizeof(int));
	*val = 2015;
	kernel<<<1, 1>>>(val);
	CUDA_SAFE_CALL(hipGetLastError());
	CUDA_SAFE_CALL(hipDeviceSynchronize());
	printf("CPU value read: %d\n", *val);

	return 0;
}

